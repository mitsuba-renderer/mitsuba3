#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>

#include <mitsuba/render/optix/matrix.cuh>
#include <mitsuba/render/optix/common.h>

// Include all shapes CUDA headers to generate their PTX programs
#include <mitsuba/render/optix/shapes.h>

extern "C" __global__ void __miss__ms() {
    optixSetPayload_0(__float_as_int(HIP_INF_F));
    optixSetPayload_4(0);
}
