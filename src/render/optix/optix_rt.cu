#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>

#include <mitsuba/render/optix/matrix.cuh>
#include <mitsuba/render/optix/common.h>

// Include all shapes CUDA headers to generate their PTX programs
#include <mitsuba/render/optix/shapes.h>

extern "C" __global__ void __miss__ms() {
    if (optixGetRayFlags() == (OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT |
                               OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT)) { // ray_test
        optixSetPayload_0(0);
    } else {
        optixSetPayload_0(__float_as_int(HIP_INF_F));
        optixSetPayload_4(0);
    }
}
